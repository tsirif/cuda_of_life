#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>

#define T_I 0
#define C_I 1
#define B_I 2
#define L_J 0
#define C_J 1
#define R_J 2

#include <stdio.h>

__device__
ext_bboard bboard_to_ext(bboard val, int m_i, int m_j) {
    ext_bboard ext = 0;
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            if (BOARD_IS_SET(val, i, j)) EXT_SET_BOARD(ext, i, j);
        }
    }
    return ext << (EXT_POS(m_i, m_j));
}

__device__
bboard reverse(bboard x) {
    x = (((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1));
    x = (((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2));
    x = (((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4));
    x = (((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8));
    return ((x >> 16) | (x << 16));

    // inline assembly way:
    //    bboard res;
    //    asm("brev.b32 %0, %1;" : "=r"(res) : "r"(x));
    //    return res;
}

__device__
bboard ext_to_bboard(ext_bboard val) {
    bboard res = 0;
    for (int i = 1; i < EXT_HEIGHT - 1; i++) {
        for (int j = 1; j < EXT_WIDTH - 1; j++) {
            if (EXT_BOARD_IS_SET(val, i, j)) SET_BOARD(res, i - 1, j - 1);
        }
    }
    return res;
}

__device__
ext_bboard gol(ext_bboard cell) {
    const ext_bboard L1 = cell >> 1;
    const ext_bboard L2 = cell << 1;
    const ext_bboard L3 = cell << EXT_WIDTH;
    const ext_bboard L4 = cell >> EXT_WIDTH;
    const ext_bboard L5 = cell << (EXT_WIDTH + 1);
    const ext_bboard L6 = cell >> (EXT_WIDTH + 1);
    const ext_bboard L7 = cell << (EXT_WIDTH - 1);
    const ext_bboard L8 = cell >> (EXT_WIDTH - 1);
    ext_bboard S0, S1, S2, S3;
    S0 = S1 = S2 = S3 = 0;

    S0 = ~(L1 | L2);
    S1 = L1 ^ L2;
    S2 = L1 & L2;

    S3 = L3 & S2;
    S2 = (S2 & ~L3) | (S1 & L3);
    S1 = (S1 & ~L3) | (S0 & L3);
    S0 = S0 & ~L3;

    S3 = (S3 & ~L4) | (S2 & L4);
    S2 = (S2 & ~L4) | (S1 & L4);
    S1 = (S1 & ~L4) | (S0 & L4);
    S0 = S0 & ~L4;

    S3 = (S3 & ~L5) | (S2 & L5);
    S2 = (S2 & ~L5) | (S1 & L5);
    S1 = (S1 & ~L5) | (S0 & L5);
    S0 = S0 & ~L5;

    S3 = (S3 & ~L6) | (S2 & L6);
    S2 = (S2 & ~L6) | (S1 & L6);
    S1 = (S1 & ~L6) | (S0 & L6);
    S0 = S0 & ~L6;

    S3 = (S3 & ~L7) | (S2 & L7);
    S2 = (S2 & ~L7) | (S1 & L7);
    S1 = (S1 & ~L7) | (S0 & L7);
    S0 = S0 & ~L7;

    S3 = (S3 & ~L8) | (S2 & L8);
    S2 = (S2 & ~L8) | (S1 & L8);

    return (((S2 & cell) | S3));
}


__global__
void calculate_next_generation_no_rem(const bboard* d_a,
                                      bboard* d_result,
                                      const int dim,
                                      const int dim_board_w,
                                      const int dim_board_h,
                                      const size_t pitch
                                     ) {

    const int major_j = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;  // col
    const int major_i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;  // row

    if ((__mul24(major_j, WIDTH) >= dim) || (__mul24(major_i, HEIGHT) >= dim)) return;

    bboard neighbors[3][3];
    {
        const int major_l = (major_j - 1 + dim_board_w) % dim_board_w;
        const int major_r = (major_j + 1) % dim_board_w;
        const int major_t = (major_i - 1 + dim_board_h) % dim_board_h;
        const int major_b = (major_i + 1) % dim_board_h;
        bboard* row_c = (bboard*)((char*)d_a + major_i * pitch);
        bboard* row_t = (bboard*)((char*)d_a + major_t* pitch);
        bboard* row_b = (bboard*)((char*)d_a + major_b * pitch);
        neighbors[C_I][C_J] = row_c[major_j];
        neighbors[C_I][L_J] = row_c[major_l] & BBOARD_RIGHT_COL_MASK;
        neighbors[C_I][R_J] = row_c[major_r] & BBOARD_LEFT_COL_MASK;
        neighbors[T_I][C_J] = row_t[major_j] & BBOARD_BOTTOM_ROW_MASK;
        neighbors[T_I][L_J] = row_t[major_l] & BBOARD_BOTTOM_ROW_MASK & BBOARD_RIGHT_COL_MASK;
        neighbors[T_I][R_J] = row_t[major_r] & BBOARD_BOTTOM_ROW_MASK & BBOARD_LEFT_COL_MASK;
        neighbors[B_I][C_J] = row_b[major_j] & BBOARD_UPPER_ROW_MASK;
        neighbors[B_I][L_J] = row_b[major_l] & BBOARD_UPPER_ROW_MASK & BBOARD_RIGHT_COL_MASK;
        neighbors[B_I][R_J] = row_b[major_r] & BBOARD_UPPER_ROW_MASK & BBOARD_LEFT_COL_MASK;

        neighbors[C_I][L_J] = (neighbors[C_I][L_J]) >> (WIDTH - 1);
        neighbors[C_I][R_J] = (neighbors[C_I][R_J]) << (WIDTH - 1);
        neighbors[T_I][C_J] = (neighbors[T_I][C_J]) >> ((HEIGHT - 1) * WIDTH);
        neighbors[T_I][L_J] = reverse(neighbors[T_I][L_J]); // corner
        neighbors[T_I][R_J] = reverse(neighbors[T_I][R_J]); // corner
        neighbors[B_I][C_J] = neighbors[B_I][C_J] << ((HEIGHT - 1) * WIDTH);
        neighbors[B_I][L_J] = reverse(neighbors[B_I][L_J]); // corner
        neighbors[B_I][R_J] = reverse(neighbors[B_I][R_J]); // corner
    }

    ext_bboard res = 0;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            res |= bboard_to_ext(neighbors[i][j], i, j);
        }
    }

    res = gol(res);

    bboard* row_result = (bboard*)((char*)d_result + major_i * pitch);
    row_result[major_j] = ext_to_bboard(res);
}
