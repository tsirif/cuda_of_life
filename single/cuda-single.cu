#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern "C" {
#include "utils.h"
}

    /* gets last cuda error and if it's not a hipSuccess
     * prints debug information on stderr and aborts */
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err), \
                    __FILE__, __LINE__); \
            exit(1); \
        } \
    } while (0)

    //TODO: change it with nvidia's function
    /* Determines the number of threads per block.
     * Returns a power of 2 number that evenly divides the total number of elements*/
    int find_thread_count(const int dim) {
        if (dim == 0) return 0;
        int result = 2;
        while ((dim % result == 0) && (result < 1024)) result *= 2;
        return result >> 1;
    }

    __global__ void cuda_compute(int* d_help, const int* d_table, int N) {
        const int cell_id = blockIdx.x * blockDim.x + threadIdx.x;
        const int j = cell_id % N;
        const int i = (cell_id - j) / N;

        const int left = (i - 1 + N) % N;
        const int right = (i + 1) % N;
        const int up = (j - 1 + N) % N;
        const int down = (j + 1) % N;

        const int alive_neighbors = d_table[POS(left , up)] +
                                    d_table[POS(left , j)] +
                                    d_table[POS(left , down)] +
                                    d_table[POS(i    , up)] +
                                    d_table[POS(i    , down)] +
                                    d_table[POS(right, up)] +
                                    d_table[POS(right, j)] +
                                    d_table[POS(right, down)] ;
        if (cell_id < N * N)
            d_help[cell_id] = (alive_neighbors == 3) || (alive_neighbors == 2 && d_table[cell_id]) ? 1 : 0;
    }

    int main(int argc, char** argv) {
        if (argc < 3) {
            printf("usage: %s FILE dimension\n", argv[0]);
            exit(1);
        }

        int n_runs;
        if (argc == 4) n_runs = atoi(argv[3]);
        else n_runs = DFL_RUNS;

        const int N = atoi(argv[2]);
        const int total_elements = N * N;
        const int mem_size = total_elements * sizeof(int);

        char* filename = argv[1];
        int* table;
        fprintf(stderr, "Reading %dx%d table from file %s\n", N, N, filename);
        table = (int*) malloc(mem_size);
        read_from_file(table, filename, N, N);
        fprintf(stderr,"Finished reading table\n");

        int t_count = find_thread_count(total_elements);
        dim3 thread_count(t_count);
        dim3 blocks_count(total_elements / t_count);

        int* d_help, *d_table;
        hipMalloc((void**) &d_help,  mem_size);
        cudaCheckErrors("malloc fail");

        hipMalloc((void**) &d_table, mem_size);
        cudaCheckErrors("malloc fail");

        hipMemcpy(d_table, table, mem_size, hipMemcpyHostToDevice);
        cudaCheckErrors("memcpy fail");

        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start) ;
        hipEventCreate(&stop) ;
        hipEventRecord(start, 0) ;

        for (int i = 0; i < n_runs; ++i) {
            cuda_compute <<< blocks_count, thread_count >>>(d_help, d_table, N);
            cudaCheckErrors("compute fail");
            swap(&d_table, &d_help);
        }

        hipEventRecord(stop, 0) ;
        hipEventSynchronize(stop) ;
        hipEventElapsedTime(&time, start, stop) ;
        printf("%f\n", time);

        hipMemcpy(table, d_table, total_elements * sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceReset();
        save_table(table, N, N, "cuda-results.bin");
    }
